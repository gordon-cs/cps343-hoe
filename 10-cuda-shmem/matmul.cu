#include "hip/hip_runtime.h"
// $Smake: nvcc -Xptxas -v -arch=sm_30 -O2 -o %F %f wtime.c

#include <cstdio>
#include <hip/hip_runtime.h>
#include "wtime.h"

#define IDX(i,j,n) ((i)*(n)+j)

#if !defined(BS)
const int BlockSize = 16;
#else
const int BlockSize = BS;  // normally 64 or less
#endif

const int MaxSizeToDisplay = 25;

typedef float FLOAT;
//typedef double FLOAT;

//----------------------------------------------------------------------------

// matrix-matrix kernel (global memory)
__global__ void matmulGlobal( FLOAT* c, FLOAT* a, FLOAT* b, int n )
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if ( col < n && row < n )
    {
	FLOAT sum = 0.0;
	for ( int k = 0; k < n; k++ )
	{
	    sum += a[IDX(row,k,n)] * b[IDX(k,col,n)];
	}
	c[IDX(row,col,n)] = sum;
    }
}

//----------------------------------------------------------------------------

// matrix-matrix kernel (shared memory)
__global__ void matmulShared( FLOAT* c, FLOAT* a, FLOAT* b, int n )
{
    // element of matrix c to compute
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // loop over blocks from block row of matrix a and
    // block column of matrix b.
    FLOAT sum = 0.0;
    int numBlocks = ( n + BlockSize - 1 ) / BlockSize;
    for ( int m = 0; m < numBlocks; m++ )
    {
	// copy block from matrix to shared memory
	__shared__ FLOAT a_s[BlockSize][BlockSize];
	__shared__ FLOAT b_s[BlockSize][BlockSize];
	int c = m * BlockSize + threadIdx.x;
	int r = m * BlockSize + threadIdx.y;
	a_s[threadIdx.y][threadIdx.x] = a[IDX(row,c,n)];
	b_s[threadIdx.y][threadIdx.x] = b[IDX(r,col,n)];
	__syncthreads();

	// length of this part of row-column product is BlockSize
	// except for last block when it may be smaller
	int sliceLen = ( m == numBlocks - 1 ? n - m * BlockSize : BlockSize );

	// compute this part of row-column product
	for ( int k = 0; k < sliceLen; k++ )
	{
	    sum += a_s[threadIdx.y][k] * b_s[k][threadIdx.x];
	}
	__syncthreads();
    }

    // all done; store computed element in matrix c
    if ( col < n && row < n ) c[IDX(row,col,n)] = sum;
}

//----------------------------------------------------------------------------

void initializeMatrix( FLOAT* a, int m, int n )
{
    for ( int i = 0; i < m; i++ )
    {
	for ( int j = 0; j < n; j++ )
	{
	    a[IDX(i,j,n)] = -1.0 * i + j;
	}
    }
}

//----------------------------------------------------------------------------

void dumpMatrix( FLOAT* a, int m, int n )
{
    for ( int i = 0; i < m; i++ )
    {
	printf( "[" );
	for ( int j = 0; j < n; j++ )
	{
	    printf( " %8.2f", a[IDX(i,j,n)] );
	}
	printf( "]\n" );
    }
}

//----------------------------------------------------------------------------

int main( int argc, char* argv[] )
{
    double t0, t1;

    int n = 4;
    if ( argc > 1 ) n = atoi( argv[1] );
    if ( n <= 0 ) n = 4; // safety check
    printf( "matrix-matrix product with %dx%d matrices.\n", n, n );

    // Declare and allocate memory for matrices
    FLOAT* a = new FLOAT [n * n];
    FLOAT* b = new FLOAT [n * n];
    FLOAT* c = new FLOAT [n * n];  // C = A * B

    // Initialize and display matrices (if small enough)
    initializeMatrix( a, n, n );
    initializeMatrix( b, n, n );
    if ( n <= MaxSizeToDisplay )
    {
	printf( "A =\n" );
	dumpMatrix( a, n, n );
	printf( "\nB =\n" );
	dumpMatrix( b, n, n );
    }

    // Declare and allocate memory for matrices on device
    size_t matrixSize = n * n * sizeof( FLOAT );
    FLOAT* a_d;  // device memory for first factor
    FLOAT* b_d;  // device memory for second factor
    FLOAT* c_d;  // device memory for product
    hipMalloc( (void**) &a_d, matrixSize );
    hipMalloc( (void**) &b_d, matrixSize );
    hipMalloc( (void**) &c_d, matrixSize );

    // Initialize matrices on device
    hipMemcpy( a_d, a, matrixSize, hipMemcpyHostToDevice );
    hipMemcpy( b_d, b, matrixSize, hipMemcpyHostToDevice );

    // Set up CUDA events for timing
    hipEvent_t event0, event1;
    hipEventCreate( &event0 );
    hipEventCreate( &event1 );

    // Prepare for kernel launches: use 2D grid
    dim3 blockDim( BlockSize, BlockSize );
    dim3 gridDim( ( n + blockDim.x - 1 ) / blockDim.x,
		  ( n + blockDim.y - 1 ) / blockDim.y ); 

    // Compute product using global-memory-only kernel
    t0 = wtime();
    hipEventRecord( event0, 0 );
    matmulGlobal<<<gridDim, blockDim>>>( c_d, a_d, b_d, n );
    hipEventRecord( event1, 0 );
    hipEventSynchronize( event1 );  // wait for event 1 to complete
    t1 = wtime();

    hipMemcpy( c, c_d, matrixSize, hipMemcpyDeviceToHost );
    if ( n <= MaxSizeToDisplay )
    {
	printf( "\n(Global Memory Only) A*B =\n" );
	dumpMatrix( c, n, n );
    }

    // Report times
    float global_time_ms;
    hipEventElapsedTime( &global_time_ms, event0, event1 );
    double global_wall_time = t1 - t0;
    printf( "Global kernel time = %e sec, elapsed wall time = %e sec\n",
	    global_time_ms / 1000.0, global_wall_time );

    // Compute product using shared-memory kernel
    t0 = wtime();
    hipEventRecord( event0, 0 );
    matmulShared<<<gridDim, blockDim>>>( c_d, a_d, b_d, n );
    hipEventRecord( event1, 0 );
    hipEventSynchronize( event1 );  // wait for event 1 to complete
    t1 = wtime();

    hipMemcpy( c, c_d, matrixSize, hipMemcpyDeviceToHost );
    if ( n <= MaxSizeToDisplay )
    {
	printf( "\n(with shared memory) A*B =\n" );
	dumpMatrix( c, n, n );
    }

    // Report times and speedup
    float shared_time_ms;
    hipEventElapsedTime( &shared_time_ms, event0, event1 );
    double shared_wall_time = t1 - t0;
    printf( "Shared kernel time = %e sec, elapsed wall time = %e sec\n",
	    shared_time_ms / 1000.0, shared_wall_time );
    printf( "Device speedup = %6.2f, Wall clock speedup = %6.2f\n",
	    global_time_ms / shared_time_ms,
	    global_wall_time / shared_wall_time );

    // all done; "let my people go!"
    hipFree( a_d );
    hipFree( b_d );
    hipFree( c_d );
    delete [] a;
    delete [] b;
    delete [] c;

    return 0;
}
