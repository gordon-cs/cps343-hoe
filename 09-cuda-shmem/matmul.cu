#include "hip/hip_runtime.h"
// $Smake: nvcc -Xptxas -v -O2 -o %F %f wtime.c
//
// Demonstrates use of device shared memory in matrix-matrix multiplication.
//
// Jonathan Senning <jonathan.senning@gordon.edu>
// Department of Mathematics and Computer Science
// Gordon College, 255 Grapevine Road, Wenham MA 01984-1899
// Spring 2016, 2018.

#include <cstdio>
#include <hip/hip_runtime.h>
#include "wtime.h"

#define IDX(i,j,n) ((i)*(n)+j)

#if !defined(BS)
const int BlockDim = 16;
#else
const int BlockDim = BS;  // normally 32 or less
#endif

const int MaxSizeToDisplay = 25;

typedef float FLOAT;
//typedef double FLOAT;

//----------------------------------------------------------------------------

// Matrix-matrix kernel (global memory)
__global__ void matmulGlobal( FLOAT* c, FLOAT* a, FLOAT* b, int n )
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if ( col < n && row < n )
    {
	FLOAT sum = 0.0;
	for ( int k = 0; k < n; k++ )
	{
	    sum += a[IDX(row,k,n)] * b[IDX(k,col,n)];
	}
	c[IDX(row,col,n)] = sum;
    }
}

//----------------------------------------------------------------------------

// Matrix-matrix kernel (shared memory)
__global__ void matmulShared( FLOAT* c, FLOAT* a, FLOAT* b, int n )
{
    // element of matrix c to compute
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // loop over blocks from block row of matrix a and
    // block column of matrix b.
    FLOAT sum = 0.0;
    int numBlocks = ( n + BlockDim - 1 ) / BlockDim;
    for ( int m = 0; m < numBlocks; m++ )
    {
	// copy block from matrix to shared memory
	__shared__ FLOAT a_s[BlockDim][BlockDim];
	__shared__ FLOAT b_s[BlockDim][BlockDim];
	int c = m * BlockDim + threadIdx.x;
	int r = m * BlockDim + threadIdx.y;
	a_s[threadIdx.y][threadIdx.x] = a[IDX(row,c,n)];
	b_s[threadIdx.y][threadIdx.x] = b[IDX(r,col,n)];
	__syncthreads();

	// length of this part of row-column product is BlockDim
	// except for last block when it may be smaller
	int sliceLen = ( m == numBlocks - 1 ? n - m * BlockDim : BlockDim );

	// compute this part of row-column product
	for ( int k = 0; k < sliceLen; k++ )
	{
	    sum += a_s[threadIdx.y][k] * b_s[k][threadIdx.x];
	}
	__syncthreads();
    }

    // all done; store computed element in matrix c
    if ( col < n && row < n ) c[IDX(row,col,n)] = sum;
}

//-----------------------------------------------------------------------------

// Check CUDA function return error code
void cudaChkErr( hipError_t code )
{
    if ( code != hipSuccess )
    {
        fprintf( stderr, "CUDA ERROR: %s\n", hipGetErrorString( code ) );
        exit( EXIT_FAILURE );
    }
}

//----------------------------------------------------------------------------

// Fill matrix with reasonable values
void initializeMatrix( FLOAT* a, int m, int n )
{
    for ( int i = 0; i < m; i++ )
    {
	for ( int j = 0; j < n; j++ )
	{
	    a[IDX(i,j,n)] = -1.0 * i + j;
	}
    }
}

//----------------------------------------------------------------------------

// Display matrix contents
void dumpMatrix( FLOAT* a, int m, int n )
{
    for ( int i = 0; i < m; i++ )
    {
	printf( "[" );
	for ( int j = 0; j < n; j++ )
	{
	    printf( " %8.2f", a[IDX(i,j,n)] );
	}
	printf( "]\n" );
    }
}

//----------------------------------------------------------------------------
//----------------------------------------------------------------------------

int main( int argc, char* argv[] )
{
    double t0, t1;       // timing variables

    // Read matrix dimension from command line
    int n = 4;
    if ( argc > 1 ) n = atoi( argv[1] );
    if ( n <= 0 ) n = 4; // safety check
    printf( "matrix-matrix product with %dx%d matrices.\n", n, n );
    printf( "BlockDim = %d\n", BlockDim );

    // Declare and allocate memory for matrices
    FLOAT* a = new FLOAT [n * n];
    FLOAT* b = new FLOAT [n * n];
    FLOAT* c = new FLOAT [n * n];  // C = A * B

    // Initialize and display matrices (if small enough)
    initializeMatrix( a, n, n );
    initializeMatrix( b, n, n );
    if ( n <= MaxSizeToDisplay )
    {
	printf( "A =\n" );
	dumpMatrix( a, n, n );
	printf( "\nB =\n" );
	dumpMatrix( b, n, n );
    }

    // Declare and allocate memory for matrices on device
    size_t matrixSize = n * n * sizeof( FLOAT );
    FLOAT* a_d;  // device memory for first factor
    FLOAT* b_d;  // device memory for second factor
    FLOAT* c_d;  // device memory for product
    cudaChkErr( hipMalloc( (void**) &a_d, matrixSize ) );
    cudaChkErr( hipMalloc( (void**) &b_d, matrixSize ) );
    cudaChkErr( hipMalloc( (void**) &c_d, matrixSize ) );

    // Initialize matrices on device
    t0 = wtime();
    cudaChkErr( hipMemcpy( a_d, a, matrixSize, hipMemcpyHostToDevice ) );
    cudaChkErr( hipMemcpy( b_d, b, matrixSize, hipMemcpyHostToDevice ) );
    t1 = wtime();
    double data_transfer_time = t1 - t0;

    // Prepare for kernel launches: use 2D grid
    dim3 blockDim( BlockDim, BlockDim );
    dim3 gridDim( ( n + blockDim.x - 1 ) / blockDim.x,
		  ( n + blockDim.y - 1 ) / blockDim.y ); 

    // Compute product using global-memory-only kernel
    t0 = wtime();
    matmulGlobal<<<gridDim, blockDim>>>( c_d, a_d, b_d, n );
    cudaChkErr( hipDeviceSynchronize() ); // wait for kernel to finish
    cudaChkErr( hipGetLastError() );      // check for any errors in kernel
    t1 = wtime();
    double global_kernel_time = t1 - t0;

    // Copy result from device to host
    t0 = wtime();
    cudaChkErr( hipMemcpy( c, c_d, matrixSize, hipMemcpyDeviceToHost ) );
    t1 = wtime();
    data_transfer_time += ( t1 - t0 );
    if ( n <= MaxSizeToDisplay )
    {
	printf( "\n(Global Memory Only) A*B =\n" );
	dumpMatrix( c, n, n );
    }

    // Compute product using shared-memory kernel
    t0 = wtime();
    matmulShared<<<gridDim, blockDim>>>( c_d, a_d, b_d, n );
    cudaChkErr( hipDeviceSynchronize() ); // wait for kernel to finish
    cudaChkErr( hipGetLastError() );      // check for any errors in kernel
    t1 = wtime();
    double shared_kernel_time = t1 - t0;

    // Copy result from device to host
    cudaChkErr( hipMemcpy( c, c_d, matrixSize, hipMemcpyDeviceToHost ) );
    if ( n <= MaxSizeToDisplay )
    {
	printf( "\n(with shared memory) A*B =\n" );
	dumpMatrix( c, n, n );
    }

    // Report times and speedup
    printf( "Data transfer time = %f sec\n", data_transfer_time );
    printf( "Global kernel time = %f sec\n", global_kernel_time );
    printf( "Shared kernel time = %f sec\n", shared_kernel_time );
    printf( "Speedup = %6.2f\n", global_kernel_time / shared_kernel_time );

    // All done; "let my people go!"
    cudaChkErr( hipFree( a_d ) );
    cudaChkErr( hipFree( b_d ) );
    cudaChkErr( hipFree( c_d ) );
    delete [] a;
    delete [] b;
    delete [] c;

    return 0;
}
