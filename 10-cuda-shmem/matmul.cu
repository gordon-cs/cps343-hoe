#include "hip/hip_runtime.h"
// $Smake: nvcc -Xptxas -v -arch=sm_30 -O2 -o %F %f wtime.c

#include <cstdio>
#include <hip/hip_runtime.h>
#include "wtime.h"

#define IDX(i,j,n) ((i)*(n)+j)

#if !defined(BS)
const int BlockSize = 16;
#else
const int BlockSize = BS;  // normally 32 or less
#endif

const int MaxSizeToDisplay = 25;

typedef float FLOAT;
//typedef double FLOAT;

//-----------------------------------------------------------------------------

void cudaChkErr(
    hipError_t code,  // value returned by CUDA runtime function
    int tag = -1     // optional tag; used to help identify call with error
    )
//
// Checks code returned by CUDA runtime function.  If not success, an error
// message is printed.  An optional second parameter is also printed if
// non-negative -- this can be used to help identify which function call
// was responsible for the error.
//
{
    if ( code != hipSuccess )
    {
        fprintf( stderr, "CUDA ERROR: %s\n", hipGetErrorString( code ) );
        if ( tag >= 0 ) fprintf( stderr, "tag = %d\n", tag );
        exit( EXIT_FAILURE );
    }
}

//----------------------------------------------------------------------------

// matrix-matrix kernel (global memory)
__global__ void matmulGlobal( FLOAT* c, FLOAT* a, FLOAT* b, int n )
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if ( col < n && row < n )
    {
	FLOAT sum = 0.0;
	for ( int k = 0; k < n; k++ )
	{
	    sum += a[IDX(row,k,n)] * b[IDX(k,col,n)];
	}
	c[IDX(row,col,n)] = sum;
    }
}

//----------------------------------------------------------------------------

// matrix-matrix kernel (shared memory)
__global__ void matmulShared( FLOAT* c, FLOAT* a, FLOAT* b, int n )
{
    // element of matrix c to compute
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // loop over blocks from block row of matrix a and
    // block column of matrix b.
    FLOAT sum = 0.0;
    int numBlocks = ( n + BlockSize - 1 ) / BlockSize;
    for ( int m = 0; m < numBlocks; m++ )
    {
	// copy block from matrix to shared memory
	__shared__ FLOAT a_s[BlockSize][BlockSize];
	__shared__ FLOAT b_s[BlockSize][BlockSize];
	int c = m * BlockSize + threadIdx.x;
	int r = m * BlockSize + threadIdx.y;
	a_s[threadIdx.y][threadIdx.x] = a[IDX(row,c,n)];
	b_s[threadIdx.y][threadIdx.x] = b[IDX(r,col,n)];
	__syncthreads();

	// length of this part of row-column product is BlockSize
	// except for last block when it may be smaller
	int sliceLen = ( m == numBlocks - 1 ? n - m * BlockSize : BlockSize );

	// compute this part of row-column product
	for ( int k = 0; k < sliceLen; k++ )
	{
	    sum += a_s[threadIdx.y][k] * b_s[k][threadIdx.x];
	}
	__syncthreads();
    }

    // all done; store computed element in matrix c
    if ( col < n && row < n ) c[IDX(row,col,n)] = sum;
}

//----------------------------------------------------------------------------

void initializeMatrix( FLOAT* a, int m, int n, double k )
{
    for ( int i = 0; i < m; i++ )
    {
	for ( int j = 0; j < n; j++ )
	{
	    a[IDX(i,j,n)] = k * ( -1.0 * i + j );// / ( n * m );
	}
    }
}

//----------------------------------------------------------------------------

void dumpMatrix( FLOAT* a, int m, int n )
{
    for ( int i = 0; i < m; i++ )
    {
	printf( "[" );
	for ( int j = 0; j < n; j++ )
	{
	    printf( " %8.2f", a[IDX(i,j,n)] );
	}
	printf( "]\n" );
    }
}

//----------------------------------------------------------------------------

int main( int argc, char* argv[] )
{
    double t0, t1;

    int n = 4;
    if ( argc > 1 ) n = atoi( argv[1] );
    if ( n <= 0 ) n = 4; // safety check
    printf( "matrix-matrix product with %dx%d matrices.\n", n, n );

    // Declare and allocate memory for matrices
    FLOAT* a = new FLOAT [n * n];
    FLOAT* b = new FLOAT [n * n];
    FLOAT* c = new FLOAT [n * n];  // C = A * B

    // Initialize and display matrices (if small enough)
    initializeMatrix( a, n, n, 0.1 );
    initializeMatrix( b, n, n, 0.01 );
    if ( n <= MaxSizeToDisplay )
    {
	printf( "A =\n" );
	dumpMatrix( a, n, n );
	printf( "\nB =\n" );
	dumpMatrix( b, n, n );
    }

    // Declare and allocate memory for matrices on device
    size_t matrixSize = n * n * sizeof( FLOAT );
    FLOAT* a_d;  // device memory for first factor
    FLOAT* b_d;  // device memory for second factor
    FLOAT* c_d;  // device memory for product
    cudaChkErr( hipMalloc( (void**) &a_d, matrixSize ) );
    cudaChkErr( hipMalloc( (void**) &b_d, matrixSize ) );
    cudaChkErr( hipMalloc( (void**) &c_d, matrixSize ) );

    // Initialize matrices on device
    cudaChkErr( hipMemcpy( a_d, a, matrixSize, hipMemcpyHostToDevice ) );
    cudaChkErr( hipMemcpy( b_d, b, matrixSize, hipMemcpyHostToDevice ) );

    // Set up CUDA events for timing
    hipEvent_t event0, event1;
    cudaChkErr( hipEventCreate( &event0 ) );
    cudaChkErr( hipEventCreate( &event1 ) );

    // Prepare for kernel launches: use 2D grid
    dim3 blockDim( BlockSize, BlockSize );
    dim3 gridDim( ( n + blockDim.x - 1 ) / blockDim.x,
		  ( n + blockDim.y - 1 ) / blockDim.y ); 

    // Compute product using global-memory-only kernel
    t0 = wtime();
    hipEventRecord( event0, 0 );
    matmulGlobal<<<gridDim, blockDim>>>( c_d, a_d, b_d, n );
    cudaChkErr( hipDeviceSynchronize() );
    cudaChkErr( hipGetLastError() );
    cudaChkErr( hipEventRecord( event1, 0 ) );
    cudaChkErr( hipEventSynchronize( event1 ) );// wait for event 1 to finish
    t1 = wtime();

    cudaChkErr( hipMemcpy( c, c_d, matrixSize, hipMemcpyDeviceToHost ) );
    if ( n <= MaxSizeToDisplay )
    {
	printf( "\n(Global Memory Only) A*B =\n" );
	dumpMatrix( c, n, n );
    }

    // Report times
    float global_time_ms;
    cudaChkErr( hipEventElapsedTime( &global_time_ms, event0, event1 ) );
    double global_wall_time = t1 - t0;
    printf( "Global kernel time = %e sec, elapsed wall time = %e sec\n",
	    global_time_ms / 1000.0, global_wall_time );

    // Compute product using shared-memory kernel
    t0 = wtime();
    cudaChkErr( hipEventRecord( event0, 0 ) );
    matmulShared<<<gridDim, blockDim>>>( c_d, a_d, b_d, n );
    cudaChkErr( hipDeviceSynchronize() );
    cudaChkErr( hipGetLastError() );
    cudaChkErr( hipEventRecord( event1, 0 ) );
    cudaChkErr( hipEventSynchronize( event1 ) );// wait for event 1 to finish
    t1 = wtime();

    cudaChkErr( hipMemcpy( c, c_d, matrixSize, hipMemcpyDeviceToHost ) );
    if ( n <= MaxSizeToDisplay )
    {
	printf( "\n(with shared memory) A*B =\n" );
	dumpMatrix( c, n, n );
    }

    double sum = 0.0;
    for ( int i = 0; i<n*n; i++ )
        sum += c[i] / double( n * n );
    printf( "sum = %f\n", sum );
    
    // Report times and speedup
    float shared_time_ms;
    cudaChkErr( hipEventElapsedTime( &shared_time_ms, event0, event1 ) );
    double shared_wall_time = t1 - t0;
    printf( "Shared kernel time = %e sec, elapsed wall time = %e sec\n",
	    shared_time_ms / 1000.0, shared_wall_time );
    printf( "Device speedup = %6.2f, Wall clock speedup = %6.2f\n",
	    global_time_ms / shared_time_ms,
	    global_wall_time / shared_wall_time );

    // all done; "let my people go!"
    cudaChkErr( hipFree( a_d ) );
    cudaChkErr( hipFree( b_d ) );
    cudaChkErr( hipFree( c_d ) );
    delete [] a;
    delete [] b;
    delete [] c;

    return 0;
}
