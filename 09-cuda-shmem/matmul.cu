#include "hip/hip_runtime.h"
// $Smake: nvcc -DBS=32 -O3 -o %F %f wtime.c
//
// Demonstrates use of device shared memory in matrix-matrix multiplication.
//
// Jonathan Senning <jonathan.senning@gordon.edu>
// Department of Mathematics and Computer Science
// Gordon College, 255 Grapevine Road, Wenham MA 01984-1899
// Spring 2016, 2018.

#include <cstdio>
#include <hip/hip_runtime.h>
#include "wtime.h"

#define IDX(i,j,n) ((i)*(n)+j) // row major

#if !defined(BS)
const int BlockDim = 16;
#else
const int BlockDim = BS;  // needs to be 32 or less
#endif

const int MaxSizeToDisplay = 25;

typedef float FLOAT;
//typedef double FLOAT;

//----------------------------------------------------------------------------

// Matrix-matrix kernel (global memory)
__global__ void matmulGlobal(FLOAT* c, FLOAT* a, FLOAT* b, int n)
{
    // element of matrix c to compute
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col >= n || row >= n) return; // nothing to do

    FLOAT sum = (FLOAT) 0.0;
    for (int k = 0; k < n; k++)
    {
        sum += a[IDX(row,k,n)] * b[IDX(k,col,n)];
    }
    c[IDX(row,col,n)] = sum;
}

//----------------------------------------------------------------------------

// Matrix-matrix kernel (shared memory)
__global__ void matmulShared(FLOAT* c, FLOAT* a, FLOAT* b, int n)
{
    __shared__ FLOAT s_a[BlockDim][BlockDim];
    __shared__ FLOAT s_b[BlockDim][BlockDim];

    // element of matrix c to compute
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // loop over row of blocks in matrix a and column of blocks
    // in matrix b; storing blocks in shared mem, 
    FLOAT sum = (FLOAT) 0.0;
    const int numBlocks = (n + BlockDim - 1) / BlockDim;
    for (int m = 0; m < numBlocks; m++)
    {
        // copy block from matrix to shared memory
        s_a[threadIdx.y][threadIdx.x] = 0.0;
        s_b[threadIdx.y][threadIdx.x] = 0.0;
        int r = m * blockDim.y + threadIdx.y;
        int c = m * blockDim.x + threadIdx.x;
        if (row < n && c < n) s_a[threadIdx.y][threadIdx.x] = a[IDX(row,c,n)];
        if (r < n && col < n) s_b[threadIdx.y][threadIdx.x] = b[IDX(r,col,n)];
        __syncthreads();

        // compute this part of row-column product
        for (int k = 0; k < BlockDim; k++)
        {
            sum += s_a[threadIdx.y][k] * s_b[k][threadIdx.x];
        }
        __syncthreads();
    }

    // all done; store computed element in matrix c
    if (row < n && col < n) c[IDX(row,col,n)] = sum;
}

//-----------------------------------------------------------------------------

// Check CUDA function return error code
void cudaChkErr(hipError_t code)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA ERROR: %s\n", hipGetErrorString(code));
        exit(EXIT_FAILURE);
    }
}

//----------------------------------------------------------------------------

// Fill matrix with reasonable values
void initializeMatrix(FLOAT* a, int m, int n, FLOAT sf = 1.0)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            a[IDX(i,j,n)] = sf * (-1.0 * i + j);
            //a[IDX(i,j,n)] = sf * ((i+1) * 10 + j+1);
        }
    }
}

//----------------------------------------------------------------------------

// Display matrix contents
void dumpMatrix(FLOAT* a, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        printf("[");
        for (int j = 0; j < n; j++)
        {
            printf(" %8.2f", a[IDX(i,j,n)]);
        }
        printf("]\n");
    }
}

//----------------------------------------------------------------------------
//----------------------------------------------------------------------------

int main(int argc, char* argv[])
{
    double t0, t1;       // timing variables

    // Read matrix dimension from command line
    int n = 4;
    if (argc > 1) n = atoi(argv[1]);
    if (n <= 0) n = 4; // safety check
    printf("Matrix-matrix product with %dx%d matrices.\n", n, n);
    printf("BlockDim = %d; Threads per block = %d\n", BlockDim,
           BlockDim * BlockDim);

    // Declare and allocate memory for matrices
    FLOAT* a = new FLOAT [n * n];
    FLOAT* b = new FLOAT [n * n];
    FLOAT* c = new FLOAT [n * n];  // C = A * B

    // Initialize and display matrices (if small enough)
    initializeMatrix(a, n, n, 1.0);
    initializeMatrix(b, n, n, 2.0);
    if (n <= MaxSizeToDisplay)
    {
        printf("A =\n");
        dumpMatrix(a, n, n);
        printf("\nB =\n");
        dumpMatrix(b, n, n);
    }

    // Declare and allocate memory for matrices on device
    size_t matrixSize = n * n * sizeof(FLOAT);
    FLOAT* d_a;  // device memory for first factor
    FLOAT* d_b;  // device memory for second factor
    FLOAT* d_c;  // device memory for product
    cudaChkErr(hipMalloc(&d_a, matrixSize));
    cudaChkErr(hipMalloc(&d_b, matrixSize));
    cudaChkErr(hipMalloc(&d_c, matrixSize));

    // Initialize matrices on device
    t0 = wtime();
    cudaChkErr(hipMemcpy(d_a, a, matrixSize, hipMemcpyHostToDevice));
    cudaChkErr(hipMemcpy(d_b, b, matrixSize, hipMemcpyHostToDevice));
    t1 = wtime();
    double data_transfer_time = t1 - t0;

    // Prepare for kernel launches: use 2D grid
    dim3 blockDim(BlockDim, BlockDim);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
                 (n + blockDim.y - 1) / blockDim.y); 

    // Compute product using global-memory-only kernel
    t0 = wtime();
    matmulGlobal<<<gridDim, blockDim>>>(d_c, d_a, d_b, n);
    cudaChkErr(hipDeviceSynchronize()); // wait for kernel to finish
    cudaChkErr(hipGetLastError());      // check for any errors in kernel
    t1 = wtime();
    double global_kernel_time = t1 - t0;

    // Copy result from device to host
    t0 = wtime();
    cudaChkErr(hipMemcpy(c, d_c, matrixSize, hipMemcpyDeviceToHost));
    t1 = wtime();
    data_transfer_time += (t1 - t0);
    if (n <= MaxSizeToDisplay)
    {
        printf("\n(Global Memory Only) A*B =\n");
        dumpMatrix(c, n, n);
    }

    // Compute product using shared-memory kernel
    t0 = wtime();
    matmulShared<<<gridDim, blockDim>>>(d_c, d_a, d_b, n);
    cudaChkErr(hipDeviceSynchronize()); // wait for kernel to finish
    cudaChkErr(hipGetLastError());      // check for any errors in kernel
    t1 = wtime();
    double shared_kernel_time = t1 - t0;

    // Copy result from device to host
    cudaChkErr(hipMemcpy(c, d_c, matrixSize, hipMemcpyDeviceToHost));
    if (n <= MaxSizeToDisplay)
    {
        printf("\n(with shared memory) A*B =\n");
        dumpMatrix(c, n, n);
    }

    // Report times and speedup
    printf("Data transfer time = %f sec\n", data_transfer_time);
    printf("Global kernel time = %f sec\n", global_kernel_time);
    printf("Shared kernel time = %f sec\n", shared_kernel_time);
    printf("Speedup = %6.2f\n", global_kernel_time / shared_kernel_time);

    // All done; "let my people go!"
    cudaChkErr(hipFree(d_a));
    cudaChkErr(hipFree(d_b));
    cudaChkErr(hipFree(d_c));
    delete [] a;
    delete [] b;
    delete [] c;

    return 0;
}
