// $Smake: nvcc -O2 -o %F %f
//
// add-vectors.cu - addition of two arrays on GPU device
//
// This program follows a very standard pattern:
//  1) allocate memory on host
//  2) allocate memory on device
//  3) initialize memory on host
//  4) copy memory from host to device
//  5) execute kernel(s) on device
//  6) copy result(s) from device to host
//
// Note: it may be possible to initialize memory directly on the device,
// in which case steps 3 and 4 are not necessary, and step 1 is only
// necessary to allocate memory to hold results.

#include <stdio.h>
#include <hip/hip_runtime.h>

//-----------------------------------------------------------------------------
// Kernel that executes on CUDA device

__global__ void add_vectors(
    float *c,      // out - pointer to result vector c
    float *a,      // in  - pointer to summand vector a
    float *b,      // in  - pointer to summand vector b
    int n          // in  - vector length
    )
{
    // Assume single block grid and 1-D block
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Only do calculation if we have real data to work with
    if (idx < n) c[idx] = a[idx] + b[idx];
}

//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
// Main program executes on host device

int main(int argc, char* argv[])
{
    // determine vector length
    int n = 10;      // set default length
    if (argc > 1)
    {
        n = atoi(argv[1]);  // override default length
        if (n <= 0)
        {
            fprintf(stderr, "Vector length must be positive\n");
            return EXIT_FAILURE;
        }
    }

    // determine vector size in bytes
    const size_t vector_size = n * sizeof(float);

    // declare pointers to vectors in host memory and allocate memory
    float *a, *b, *c;
    a = (float*) malloc(vector_size);
    b = (float*) malloc(vector_size);
    c = (float*) malloc(vector_size);

    // declare pointers to vectors in device memory and allocate memory
    float *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a, vector_size);
    hipMalloc((void**) &d_b, vector_size);
    hipMalloc((void**) &d_c, vector_size);

    // initialize vectors and copy them to device
    for (int i = 0; i < n; i++)
    {
        a[i] =   1.0 * i;
        b[i] = 100.0 * i;
    }
    hipMemcpy(d_a, a, vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, vector_size, hipMemcpyHostToDevice);

    // do calculation on device
    int block_size = 1024;
    int num_blocks = (n - 1 + block_size) / block_size;
    add_vectors<<<num_blocks, block_size>>>(d_c, d_a, d_b, n);

    // retrieve result from device and store on host
    hipMemcpy(c, d_c, vector_size, hipMemcpyDeviceToHost);

    // print results for vectors up to length 100
    if (n <= 100)
    {
        for (int i = 0; i < n; i++)
        {
            printf("%8.2f + %8.2f = %8.2f\n", a[i], b[i], c[i]);
        }
    }

    // cleanup and quit
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
  
    return 0;
}
